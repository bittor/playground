#include "hip/hip_runtime.h"
__global__ bitonic()
{
	const auto tIdx = threadIdx.x;
	const int array_size = 1024;
	__share__ int array[];

	for(int stage = 1; stage <= 10; stage++)
	{
		const int seq = (int)powf(2f,(float)stage);
		const thd_in_seq = seq/2;
		const int seqIdx = tIdx / thd_in_seq;
		const int idx_in_seq = tIdx % thd_in_seq;
		bool orient = seqIdx & 0x1;// 0:desencding
		const int base = seqIdx * seq;
		for(int step = 1; step <= stage; step++)
		{
			const int step_size = seq / pwdf(2f, (float)(step-1));
			const int thd_in_step = thd_in_seq / pwdf(2f,(float)(step-1));
			const int groupIdx = idx_in_seq / thd_in_step;
			cosnt int idx_in_step = idx_in_seq % thd_in_step;
			const int distance_swap = seq / (int)powf(2f,(float)step);
			const after_base = base + groupIdx * step_size + idx_in_step;
			
			swap(array[after_base], array[after_base+distance_swap])

		}


	}




}
